#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <time.h>

/**
* Michael O'Brien 20250121
* michael at obrienlabs.dev
* 128 bit version
* Mandelbrot set on NVidia GPUs like the RTX-3500 ada,RTX-A4000,RTX-A4500,RTX-4090 ada and RTX-A6000
* https://github.com/ObrienlabsDev/performance
* https://github.com/ObrienlabsDev/fractals
*
* https://docs.nvidia.com/cuda/floating-point/index.html 
*/
hipError_t cudaFacade(double *c, double *a, double *b, unsigned int size);


__device__ uint32_t mandel_double(double cr, double ci, int max_iter) {
    double zr = 0;
    double zi = 0;
    double zrsqr = 0;
    double zisqr = 0;

    uint32_t i;

    for (i = 0; i < max_iter; i++) {
        zi = zr * zi;
        zi += zi;
        zi += ci;
        zr = zrsqr - zisqr + cr;
        zrsqr = zr * zr;
        zisqr = zi * zi;

        //the fewer iterations it takes to diverge, the farther from the set
        if (zrsqr + zisqr > 4.0) break;
    }

    return i;
}

__global__ void mandel_kernel(uint32_t* counts, double xmin, double ymin,
    double step, int max_iter, int dim, uint32_t* colors) {
    int pix_per_thread = dim * dim / (gridDim.x * blockDim.x);
    int tId = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = pix_per_thread * tId;
    for (int i = offset; i < offset + pix_per_thread; i++) {
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x] = colors[mandel_double(cr, ci, max_iter)];
    }
    if (gridDim.x * blockDim.x * pix_per_thread < dim * dim
        && tId < (dim * dim) - (blockDim.x * gridDim.x)) {
        int i = blockDim.x * gridDim.x * pix_per_thread + tId;
        int x = i % dim;
        int y = i / dim;
        double cr = xmin + x * step;
        double ci = ymin + y * step;
        counts[y * dim + x] = colors[mandel_double(cr, ci, max_iter)];
    }
}

__global__ void addKernel(double *c, double *a, double *b)
{
    int x = threadIdx.x;

    double zr = 0;
    double zi = 0;
    double zrsqr = 0;
    double zisqr = 0;
    int max_iter = 2000;
    double ci = -0.59990625;// 0;
    double cr = 0.4290703125; //0;

    uint32_t i;

    for (i = 0; i < max_iter; i++) {
        zi = zr * zi;
        zi += zi;
        zi += ci;
        zr = zrsqr - zisqr + cr;
        zrsqr = zr * zr;
        zisqr = zi * zi;

        if (zrsqr + zisqr > 4.0) break;
    }
    c[x] = i;// zrsqr + zisqr;
}

void singleGPUMandelbrot() {
    int deviceCount = 0;
    int dualDevice = 0;
    hipGetDeviceCount(&deviceCount);
    printf("%d CUDA devices found - reallocating\n", deviceCount);
    if (deviceCount > 1) {
        dualDevice = 1;
    }

    const int arraySize = 5;
    double a[arraySize] = { 1.0, 2.0, 3.0, 4.0, 5.0 };
    double b[arraySize] = { 10.0, 20.0, 30.0, 40.0, 50.0 };
    double c[arraySize] = { 0.0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = cudaFacade(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%lf,%lf,%lf,%lf,%lf}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return;
    }
}


hipError_t cudaFacade(double* c, double* a, double* b, unsigned int size)
{
    /*
    double cr = .9;
    double ci = .1;
    int max_iter = 2000;
    int steps = 10000;
    int threads = 512;
    int blocks = (10000 + threads - 1) / threads;  // ensure threads*blocks ≥ steps
    uint32_t* counts;
    double xmin = -2.0;
    double ymin = -2.0;
    int dim = 256;
    double step = (xmin * 2) / dim;
    uint32_t* colors;
    for (int i = 0; i < max_iter; i++) {
        colors[i] = i;
    }
    //thrust::device_vector<uint32_t> dsums(steps);         // GPU buffer
   // uint32_t* dptr = thrust::raw_pointer_cast(&dsums[0]); // get pointer
    //double* dsums = 0;
    hipError_t cudaStatus;

    
    cudaStatus = hipMalloc((void**)&counts, size * sizeof(double));
    //cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);

    mandel_kernel << <blocks, threads >> > (counts, xmin, ymin, step, max_iter, dim, colors);
    //uint32_t mandel = mandel_double(cr, ci, max_iter);

    cudaStatus = hipDeviceSynchronize();

    cudaStatus = hipMemcpy(c, counts, size * sizeof(double), hipMemcpyDeviceToHost);

    double mandel = counts[0];

    printf("real %.8f, imag %.8f, value %d\n",
        cr, ci, mandel);

    hipFree(counts);

    //return 0;
    // 
    */

    double* dev_a = 0;
    double* dev_b = 0;
    double* dev_c = 0;
    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

#define WIDTH  4096//1920
#define HEIGHT 4096//1080

#define MAX_ITER 8192

#define X_MIN -2.0f
#define X_MAX  1.0f
#define Y_MIN -1.2f
#define Y_MAX  1.2f

__global__ void mandelbrotKernel(unsigned char* output, int width, int height,
    float xMin, float xMax, float yMin, float yMax, int maxIter)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= width || py >= height) return;

    float dx = (xMax - xMin) / (float)width;
    float dy = (yMax - yMin) / (float)height;
    float x0 = xMin + px * dx;
    float y0 = yMin + py * dy;

    float x = 0.0f, y = 0.0f;
    int iter = 0;
    while ((x * x + y * y <= 4.0f) && (iter < maxIter)) {
        float xTemp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = xTemp;
        iter++;
    }

    unsigned char color = (unsigned char)(255.0f * (float)iter / (float)maxIter);
    int index = py * width + px;
    output[index] = color;
}

int main(int argc, char* argv[])
{
    int gpu = (argc > 1) ? atoi(argv[1]) : 0; // get command
    int iterations = (argc > 1) ? atoi(argv[2]) : 5000;
    printf("Using GPU #: %d for iterations: %d\n", gpu, iterations);
    hipSetDevice(gpu);
    //singleGPUMandelbrot();

    time_t timeStart, timeEnd;
    double timeElapsed;
    time(&timeStart);

    size_t imageSize = WIDTH * HEIGHT * sizeof(unsigned char);
    unsigned char* h_image = (unsigned char*)malloc(imageSize);
    unsigned char* d_image;
    hipMalloc((void**)&d_image, imageSize);

    dim3 blockSize(16, 16);
    dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x,(HEIGHT + blockSize.y - 1) / blockSize.y);

    for (int run = 0; run < iterations; run++) {
        mandelbrotKernel << <gridSize, blockSize >> > (d_image, WIDTH, HEIGHT, X_MIN, X_MAX, Y_MIN, Y_MAX, MAX_ITER);
        hipDeviceSynchronize();
        //printf("Completed %d\n", run);
        hipMemcpy(h_image, d_image, imageSize, hipMemcpyDeviceToHost);
    }
    hipFree(d_image);
    free(h_image);
    
    time(&timeEnd);
    timeElapsed = difftime(timeEnd, timeStart);

    printf("duration: %.f\n", timeElapsed);
    printf("time / run : %f\n", timeElapsed / iterations);
    
    return 0;
}
